#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>

#include "dmat.h"

#define NUM_THREADS 32

// Space for the vector data
__constant__ double * distance_vg_a_d;
__constant__ double * distance_vg_ul_d;

// Space for the resulting distance
__device__ double * distance_d_d;

void checkCudaError(const char * msg) {
	hipError_t err = hipGetLastError();
	if(hipSuccess != err) {
		if(msg != NULL) {
			printf("[WARNING] %s\n", msg);
		}
		printf("[ERROR] %s : %s\n", msg, hipGetErrorString(err));
		exit(1);
	}
}

__global__ void euclidean_kernel_same(	const double * vg_a, size_t pitch_a, size_t n_a, 
										const double * vg_ul, size_t k,
										double * d, size_t pitch_d )
{
	size_t x = blockIdx.x, y = blockIdx.y;

	if((x == y) && (x < n_a) && (threadIdx.x == 0))
		d[y * pitch_d + x] = 0.0;
  
	// If all element is to be computed
	if(y < n_a && x < y) {
		__shared__ double temp[NUM_THREADS];    

		temp[threadIdx.x] = 0.0;
    
		for(size_t offset = threadIdx.x; offset < k; offset += NUM_THREADS) {
			double t = abs(vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset]) / vg_ul[offset];
			temp[threadIdx.x] += (t * t);
		}
    
		// Sync with other threads
		__syncthreads();
    
		// Reduce
		for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
			if(threadIdx.x < stride)
				temp[threadIdx.x] += temp[threadIdx.x + stride];
			__syncthreads();
		}
	    
		// Write to global memory
		if(threadIdx.x == 0) {
			double s = sqrt(temp[0]);
			d[y * pitch_d + x] = s;
			d[x * pitch_d + y] = s;
		}
	}
}

void distance_device(	const double * vg_a_d, size_t pitch_a, size_t n_a,  
						const double * vg_ul_d, size_t k,
						double * d_d, size_t pitch_d) {

	dim3 block(NUM_THREADS, 1, 1);
	dim3 grid(n_a, n_a, 1);

	size_t fbytes = sizeof(double);

	pitch_a /= fbytes;
	pitch_d /= fbytes;

	euclidean_kernel_same<<<grid, block>>>(	vg_a_d, pitch_a, n_a,
											vg_ul_d, k,
											d_d, pitch_d);
}

void distanceGPU(	const double * vg_a, size_t pitch_a, size_t n_a, 
				const double * vg_ul, size_t k, 
				double * d, size_t pitch_d ) {
	
	size_t pitch_a_d, pitch_d_d;
	
	// Allocate space for the vectors and distances on the gpu
	hipMallocPitch((void**)&distance_vg_a_d, &pitch_a_d, k * sizeof(double), n_a);
	hipMemcpy2D(distance_vg_a_d, pitch_a_d, vg_a, pitch_a, k * sizeof(double), n_a, hipMemcpyHostToDevice);

	hipMalloc((void**)&distance_vg_ul_d, k * sizeof(double));
	hipMemcpy(distance_vg_ul_d, vg_ul, k * sizeof(double), hipMemcpyHostToDevice);
	
	hipMallocPitch((void**)&distance_d_d, &pitch_d_d, n_a * sizeof(double), n_a);
	
	checkCudaError("distance function : malloc and memcpy");
    
	distance_device(distance_vg_a_d, pitch_a_d, n_a,  
					distance_vg_ul_d, k,
					distance_d_d, pitch_d_d);
	
	checkCudaError("distance function : kernel invocation");

	// Copy the result back to cpu land now that gpu work is done
	hipMemcpy2D(d, pitch_d, distance_d_d, pitch_d_d, n_a * sizeof(double), n_a, hipMemcpyDeviceToHost);
	checkCudaError("distance function : memcpy");
    
	// Free allocated space
	hipFree(distance_vg_a_d);
	hipFree(distance_vg_ul_d);
	hipFree(distance_d_d);
}